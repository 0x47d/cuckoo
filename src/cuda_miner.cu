// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2015 John Tromp

// The edge=trimming time-memory trade-off is due to Dave Anderson:
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html


#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>
#include <openssl/sha.h>

// proof-of-work parameters
#ifndef SIZESHIFT 
#define SIZESHIFT 25
#endif
#ifndef PROOFSIZE
#define PROOFSIZE 42
#endif

#define SIZE (1UL<<SIZESHIFT)
#define HALFSIZE (SIZE/2)
#define NODEMASK (HALFSIZE-1)

typedef uint32_t u32;
typedef uint64_t u64;
#if SIZESHIFT < 32
typedef u32 nonce_t;
typedef u32 node_t;
#else
typedef u64 nonce_t;
typedef u64 node_t;
#endif

typedef struct {
  u64 v[4];
} siphash_ctx;
 
#define U8TO64_LE(p) \
  (((u64)((p)[0])      ) | ((u64)((p)[1]) <<  8) | \
   ((u64)((p)[2]) << 16) | ((u64)((p)[3]) << 24) | \
   ((u64)((p)[4]) << 32) | ((u64)((p)[5]) << 40) | \
   ((u64)((p)[6]) << 48) | ((u64)((p)[7]) << 56))
 
// derive siphash key from header
void setheader(siphash_ctx *ctx, const char *header) {
  unsigned char hdrkey[32];
  SHA256((unsigned char *)header, strlen(header), hdrkey);
  u64 k0 = U8TO64_LE(hdrkey);
  u64 k1 = U8TO64_LE(hdrkey+8);
  ctx->v[0] = k0 ^ 0x736f6d6570736575ULL;
  ctx->v[1] = k1 ^ 0x646f72616e646f6dULL;
  ctx->v[2] = k0 ^ 0x6c7967656e657261ULL;
  ctx->v[3] = k1 ^ 0x7465646279746573ULL;
}

#define ROTL(x,b) (u64)( ((x) << (b)) | ( (x) >> (64 - (b))) )
#define SIPROUND \
  do { \
    v0 += v1; v2 += v3; v1 = ROTL(v1,13); \
    v3 = ROTL(v3,16); v1 ^= v0; v3 ^= v2; \
    v0 = ROTL(v0,32); v2 += v1; v0 += v3; \
    v1 = ROTL(v1,17);   v3 = ROTL(v3,21); \
    v1 ^= v2; v3 ^= v0; v2 = ROTL(v2,32); \
  } while(0)
 
// generate edge endpoint in cuckoo graph
__device__ node_t sipnode(siphash_ctx *ctx, nonce_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = ctx->v[0], v1 = ctx->v[1], v2 = ctx->v[2], v3 = ctx->v[3] ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & NODEMASK;
}

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
typedef u32 au32;
typedef u64 au64;
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (SIZE >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// SIZE >> IDXSHIFT == HALFSIZE >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif
// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (SIZESHIFT/3))

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u32 *bits;
  __device__ void reset(nonce_t n) {
    bits[n/32] |= 1 << (n%32);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/32] >> (n%32)) & 1);
  }
  __device__ u32 block(node_t n) const {
    return ~bits[n/32];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (HALFSIZE >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  au32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(au32));
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (SIZE >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by SIZESHIFT
#define KEYBITS (64-SIZESHIFT)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  au64 *cuckoo;

  cuckoo_hash() {
    assert(cuckoo = (au64 *)calloc(CUCKOO_SIZE, sizeof(au64)));
  }
  ~cuckoo_hash() {
    free(cuckoo);
  }
  void set(node_t u, node_t v) {
    u64 niew = (u64)u << SIZESHIFT | v;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 old = 0;
      if (cuckoo[ui].compare_exchange_strong(old, niew, std::memory_order_relaxed))
        return;
      if ((old >> SIZESHIFT) == (u & KEYMASK)) {
        cuckoo[ui].store(niew, std::memory_order_relaxed);
#else
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> SIZESHIFT) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
#endif
        return;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
#ifdef ATOMIC
      u64 cu = cuckoo[ui].load(std::memory_order_relaxed);
#else
      u64 cu = cuckoo[ui];
#endif
      if (!cu)
        return 0;
      if ((cu >> SIZESHIFT) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & (SIZE-1));
      }
    }
  }
};

class cuckoo_ctx {
public:
  siphash_ctx sip_ctx;
  shrinkingset alive;
  twice_set nonleaf;
  int nthreads;

  cuckoo_ctx(const char* header, u32 n_threads) {
    setheader(&sip_ctx, header);
    nthreads = n_threads;
  }
};

#define LOGNBUCKETS	0
#define NBUCKETS	1
#define BUCKETSHIFT	(SIZESHIFT-1)
#define NONCESHIFT	(SIZESHIFT-1 - PART_BITS)
#define NODEPARTMASK	(NODEMASK >> PART_BITS)
#define NONCETRUNC	(1L << (64 - NONCESHIFT))

__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (nonce_t block = id*32; block < HALFSIZE; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (nonce_t nonce = block; alive32; alive32>>=1, nonce++) {
      if (alive32 & 1) {
        node_t u = sipnode(&ctx->sip_ctx, nonce, uorv);
        if ((u & PART_MASK) == part) {
          nonleaf.set(u >> PART_BITS);
        }
      }
    }
  }
}

__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (nonce_t block = id*32; block < HALFSIZE; block += ctx->nthreads*32) {
    u32 alive32 = alive.block(block);
    for (nonce_t nonce = block; alive32; alive32>>=1, nonce++) {
      if (alive32 & 1) {
        node_t u = sipnode(&ctx->sip_ctx, nonce, uorv);
        if ((u & PART_MASK) == part) {
          if (!nonleaf.test(u >> PART_BITS)) {
            alive.reset(nonce);
          }
        }
      }
    }
  }
}

#if 0
u32 path(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (++nu >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      pthread_exit(NULL);
    }
    us[nu] = u;
  }
  return nu;
}

typedef std::pair<node_t,node_t> edge;

void solution(cuckoo_ctx *ctx, node_t *us, u32 nu, node_t *vs, u32 nv) {
  std::set<edge> cycle;
  u32 n;
  cycle.insert(edge(*us, *vs));
  while (nu--)
    cycle.insert(edge(us[(nu+1)&~1], us[nu|1])); // u's in even position; v's in odd
  while (nv--)
    cycle.insert(edge(vs[nv|1], vs[(nv+1)&~1])); // u's in odd position; v's in even
  for (nonce_t nonce = n = 0; nonce < HALFSIZE; nonce++)
    if (ctx->alive.test(nonce)) {
      edge e(sipnode(&ctx->sip_ctx, nonce, 0), HALFSIZE+sipnode(&ctx->sip_ctx, nonce, 1));
      if (cycle.find(e) != cycle.end()) {
        printf(" %lx", (long)nonce);
        if (PROOFSIZE > 2)
          cycle.erase(e);
      }
    }
  assert(n==PROOFSIZE);
  printf("\n");
}
#endif

#include <unistd.h>

int main(int argc, char **argv) {
  int nthreads = 1;
  int maxsols  = 8;
  int ntrims   = 1 + (PART_BITS+3)*(PART_BITS+4)/2;
  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:m:n:t:")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'm':
        maxsols = atoi(optarg);
        break;
      case 'n':
        ntrims = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
    }
  }
  printf("Looking for %d-cycle on cuckoo%d(\"%s\") with 50%% edges, %d trims, %d threads\n",
               PROOFSIZE, SIZESHIFT, header, ntrims, nthreads);
  u64 edgeBytes = HALFSIZE/8, nodeBytes = TWICE_WORDS*sizeof(u32);

  cuckoo_ctx ctx(header, nthreads);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  printf("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));
  hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);

  for (u32 round=0; round < ntrims; round++) {
    for (u32 uorv = 0; uorv < 2; uorv++) {
      for (u32 part = 0; part <= PART_MASK; part++) {
        checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
        count_node_deg<<<nthreads,1>>>(device_ctx,uorv,part);
        kill_leaf_edges<<<nthreads,1>>>(device_ctx,uorv,part);
        printf("round %d part %c%d\n", round, "UV"[uorv], part);
      }
    }
  }

  u32 *bits;
  assert(bits = (u32 *)calloc(HALFSIZE/32, sizeof(u32)));
  hipMemcpy(bits, ctx.alive.bits, (HALFSIZE/32) * sizeof(u32), hipMemcpyDeviceToHost);
  checkCudaErrors(hipFree(ctx.alive.bits));
  checkCudaErrors(hipFree(ctx.nonleaf.bits));

  u32 cnt = 0;
  for (nonce_t nonce = 0; nonce < HALFSIZE; nonce++)
    cnt += ((bits[nonce/32] >> (nonce%32)) & 1) ^ 1;

  u32 load = (u32)(100L * cnt / CUCKOO_SIZE);
  printf("final load %d%%\n", load);

#if 0
  if (tp->id == 0) {
    load = (u32)(100 * alive.count() / CUCKOO_SIZE);
    if (load >= 90) {
      printf("overloaded! exiting...");
      exit(0);
    }
    delete ctx->nonleaf;
    ctx->nonleaf = 0;
    ctx->cuckoo = new cuckoo_hash();
  }
  barrier(&ctx->barry);
  cuckoo_hash &cuckoo = *ctx->cuckoo;
  node_t us[MAXPATHLEN], vs[MAXPATHLEN];
  for (nonce_t block = tp->id*32; block < HALFSIZE; block += ctx->nthreads*32) {
    for (nonce_t nonce = block; nonce < block+32 && nonce < HALFSIZE; nonce++) {
      if (alive.test(nonce)) {
        node_t u0 = sipnode(ctx, nonce, 0), v0 = sipnode(ctx, nonce, 1) + HALFSIZE;  // make v's different from u's
        node_t u = cuckoo[us[0] = u0], v = cuckoo[vs[0] = v0];
        u32 nu = path(cuckoo, u, us), nv = path(cuckoo, v, vs);
        if (us[nu] == vs[nv]) {
          u32 min = nu < nv ? nu : nv;
          for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
          u32 len = nu + nv + 1;
          printf("% 4d-cycle found at %d:%d%%\n", len, tp->id, (u32)(nonce*100L/HALFSIZE));
          if (len == PROOFSIZE && ctx->nsols < ctx->maxsols)
            solution(ctx, us, nu, vs, nv);
          continue;
        }
        if (nu < nv) {
          while (nu--)
            cuckoo.set(us[nu+1], us[nu]);
          cuckoo.set(u0, v0);
        } else {
          while (nv--)
            cuckoo.set(vs[nv+1], vs[nv]);
          cuckoo.set(v0, u0);
        }
      }
    }
  }

  thread_ctx *threads = (thread_ctx *)calloc(nthreads, sizeof(thread_ctx));
  assert(threads);
  for (int t = 0; t < nthreads; t++) {
    threads[t].id = t;
    threads[t].ctx = &ctx;
    assert(pthread_create(&threads[t].thread, NULL, worker, (void *)&threads[t]) == 0);
  }
  for (int t = 0; t < nthreads; t++)
    assert(pthread_join(threads[t].thread, NULL) == 0);
  free(threads);

  for (unsigned s = 0; s < ctx.nsols; s++) {
    printf("Solution");
    for (int i = 0; i < PROOFSIZE; i++)
      printf(" %lx", (long)ctx.sols[s][i]);
    printf("\n");
  }
#endif
  return 0;
}
